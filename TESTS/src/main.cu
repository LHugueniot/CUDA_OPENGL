#include "hip/hip_runtime.h"
#include "gtest/gtest.h"

#include <filesystem>
#include <map>
#include <sstream>
#include <string>
#include <vector>

#include "Geometry/Geometry.cuh"
#include "Geometry/LoadGeometry.cuh"
#include "Sim/PBD/PBDSolver.cuh"
#include "Viewer/Camera.h"

// Include GLM
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/type_ptr.hpp>

#define STR(s) #s
#define XSTR(s) STR(s)

static constexpr char *kAssetDirectory = XSTR(ASSETS_DIRECTORY);

template <typename T, typename P = float>
bool _EXPECT_NEAR(T a, T b, P p)
{
    EXPECT_NEAR(a, b, p);
    T diff = std::abs(a - b);
    return p > diff && diff >= 0;
}

template <typename T, typename P = float>
void EXPECT_NEAR_STDVEC(T a, T b, P p = 0.00001)
{
    ASSERT_EQ(a.size(), b.size());
    bool val = true;
    for (int i = 0; i < a.size(); i++)
    {
        val &= _EXPECT_NEAR(a.data()[i], b.data()[i], p);
    }
    if (!val)
    {
        EXPECT_EQ(a, b);
    }
}

template <typename T, typename P = float>
void EXPECT_NEAR_MATRIX(T a, T b, P p = 0.00001)
{
    ASSERT_EQ(a.rows(), b.rows());
    ASSERT_EQ(a.cols(), b.cols());
    bool val = true;
    for (int i = 0; i < a.rows() * a.cols(); i++)
    {
        val &= _EXPECT_NEAR(a.data()[i], b.data()[i], p);
    }
    if (!val)
    {
        EXPECT_EQ(a, b);
    }
}

void print_glm_mat4(glm::mat4 &mat)
{
    for (int r = 0; r < 4; r++)
    {
        for (int c = 0; c < 4; c++)
        {
            fprintf(stdout, "%7.1f", mat[r][c]);
        }
        fprintf(stdout, "\n");
    }
}

template <typename T>
std::vector<T> deviceToContainer(T *d_ptr, size_t nElems)
{
    std::vector<T> result(nElems);

    cutilSafeCall(hipMemcpy(result.data(),
                             d_ptr,
                             nElems * sizeof(T),
                             hipMemcpyDeviceToHost));
    return result;
}

float length(ei::Vector3f const &v)
{
    return sqrt(v.x() * v.x() + v.y() * v.y() + v.z() * v.z());
}

TEST(EigenTests, ProjectionAndViewTest)
{
    float radians_fov = 45.0f;
    float windowWidth = 1920.f;
    float windowHeight = 1080.f;
    float fov = 45.f;
    float far = 100.0f;
    float near = 0.1f;

    glm::mat4 glm_projection = glm::perspective(
        glm::radians(radians_fov), windowWidth / windowHeight, near, far);
    glm::mat4 glm_view = glm::lookAt(
        glm::vec3(0, 0, 30), // Camera is at (4,3,-3), in World Space
        glm::vec3(0, 0, 0),  // and looks at the origin
        glm::vec3(0, 1, 0)   // Head is up (set to 0,-1,0 to look upside-down)
    );

    glm::mat4 glm_PV = glm_projection * glm_view;

    ei::Matrix4f ei_projection;
    ei::Matrix4f ei_view;

    ei_utils::setProjMat(ei_projection, windowWidth, windowHeight, TO_RAD(fov), far, near);
    ei_utils::setLookAt(ei_view, {0, 0, 30}, {0, 0, 0}, {0, 1, 0});

    // ei::Matrix4f ei_PV = ei_projection.transpose() * ei_view.transpose();
    ei::Matrix4f ei_PV = ei_projection * ei_view;

    ASSERT_EQ(TO_RAD(fov), glm::radians(radians_fov));

    for (int i = 0; i < 16; i++)
    {
        EXPECT_NEAR(glm::value_ptr(glm_projection)[i], ei_projection.data()[i], 0.1);
        EXPECT_NEAR(glm::value_ptr(glm_view)[i], ei_view.data()[i], 0.1);
        EXPECT_NEAR(glm::value_ptr(glm_PV)[i], ei_PV.data()[i], 0.1);
    }

    //    std::cout << "PROJECTION MATRICES" << std::endl;
    //    std::cout << ei_projection << std::endl;
    //    print_glm_mat4(glm_projection);
    //
    //    std::cout << "VIEW MATRICES" << std::endl;
    //    std::cout << ei_view << std::endl;
    //    print_glm_mat4(glm_view);
    //
    //    std::cout << "PROJECTION VIEW MATRICES" << std::endl;
    //    std::cout << ei_PV.transpose() << std::endl;
    //    print_glm_mat4(glm_PV);
    //    //std::cout << glm::to_string(glm_projection) << std::endl;
    //
    //
    //    for (int i=0; i<16; i++)
    //    {
    //        auto* p = &glm_projection[0][0];
    //        assert(p[i] == ei_projection.data()[i]);
    //        auto* v = &glm_view[0][0];
    //        assert(v[i] == ei_view.data()[i]);
    //        auto* pv = &glm_PV[0][0];
    //        assert(pv[i] == ei_PV.data()[i]);
    //    }
    //
    //    for (int r=0; r<4; r++)
    //        for (int c=0; c<4; c++)
    //        {
    //            assert(glm_projection[r][c] == ei_projection(r, c));
    //            assert(glm_view[r][c] == ei_view(r, c));
    //            assert(glm_PV[r][c] == ei_PV(r, c));
    //        }
}

using MapVector3f = Eigen::Map<ei::Vector3f>;

void setupTestGeom(Geometry &testGeom,
                   std::vector<float> &vertexData)
{
    testGeom.d_nVertexPositionBufferElems = vertexData.size();

    size_t bufferBytesSize = testGeom.d_nVertexPositionBufferElems *
                             sizeof(float);

    hipMalloc((void **)&testGeom.d_vertexPositionBufferData,
               bufferBytesSize);

    hipMemcpy(testGeom.d_vertexPositionBufferData,
               &vertexData.data()[0],
               bufferBytesSize,
               hipMemcpyHostToDevice);
}

void retrieveProcessedGeom(Geometry &testGeom,
                           std::vector<float> &vertexData)
{
    vertexData.resize(testGeom.d_nVertexPositionBufferElems);
    size_t bufferBytesSize = testGeom.d_nVertexPositionBufferElems *
                             sizeof(float);
    hipMemcpy(static_cast<void *>(&vertexData.data()[0]),
               static_cast<const void *>(testGeom.d_vertexPositionBufferData),
               bufferBytesSize,
               hipMemcpyDeviceToHost);

    hipFree(testGeom.d_vertexPositionBufferData);
}

TEST(Geometry, scaleGeom1)
{
    auto testGeom = Geometry();
    std::vector<float> vertexData =
        {
            0, 0, 0,
            1, 0, 0,
            0, 1, 0,
            0, 0, 1};

    ei::Vector3f scale = {2.5f, 5.0f, 10.0f};
    setupTestGeom(testGeom, vertexData);

    scaleGeom(testGeom, scale);

    retrieveProcessedGeom(testGeom, vertexData);

    std::vector<float> expectedVertexData =
        {
            0, 0, 0,
            2.5f, 0, 0,
            0, 5.0f, 0,
            0, 0, 10.0f};

    ASSERT_EQ(vertexData, expectedVertexData);
}

TEST(Geometry, scaleGeom2)
{
    auto testGeom = Geometry();
    std::vector<float> vertexData =
        {
            0, 0, 0,
            1, 0, 0,
            0, 1, 0,
            0, 0, 1};

    ei::Vector3f scale = {5.f, 5.f, 5.f};
    setupTestGeom(testGeom, vertexData);

    scaleGeom(testGeom, scale);

    retrieveProcessedGeom(testGeom, vertexData);

    std::vector<float> expectedVertexData =
        {
            0, 0, 0,
            5.f, 0, 0,
            0, 5.f, 0,
            0, 0, 5.f};

    ASSERT_EQ(vertexData, expectedVertexData);
}

TEST(Geometry, scaleGeom3)
{
    auto testGeom = Geometry();
    std::vector<float> vertexData =
        {
            0, 0, 0,
            10.f, 0, 0,
            0, 10.f, 0,
            0, 0, 10.f};

    ei::Vector3f scale = {2.f, 2.f, 2.f};
    ei::Vector3f pivot = {5.f, 5.f, 5.f};
    setupTestGeom(testGeom, vertexData);

    scaleGeom(testGeom, scale, pivot);

    retrieveProcessedGeom(testGeom, vertexData);

    std::vector<float> expectedVertexData =
        {
            -5, -5, -5,
            15, -5, -5,
            -5, 15, -5,
            -5, -5, 15};

    ASSERT_EQ(vertexData, expectedVertexData);
}

TEST(Geometry, rotateGeom1)
{
    auto testGeom = Geometry();
    std::vector<float> vertexData =
        {
            0, 0, 0,
            10.f, 0, 0,
            0, 10.f, 0,
            0, 0, 10.f};

    ei::Vector3f axis = {0, 1.f, 0};

    setupTestGeom(testGeom, vertexData);

    rotateGeom(testGeom, axis, TO_RAD(90), {0, 0, 0});

    retrieveProcessedGeom(testGeom, vertexData);

    std::vector<float> expectedVertexData =
        {
            0, 0, 0,
            0, 0, 10.f,
            0, 10.f, 0,
            -10.f, 0, 0};

    EXPECT_NEAR_STDVEC(vertexData, expectedVertexData);
}

TEST(Geometry, rotateGeom2)
{
    auto testGeom = Geometry();
    std::vector<float> vertexData =
        {
            0, 0, 0,
            10.f, 0, 0,
            0, 10.f, 0,
            0, 0, 10.f};

    ei::Vector3f axis = {0, 1.f, 0};

    setupTestGeom(testGeom, vertexData);

    rotateGeom(testGeom, axis, TO_RAD(90), {5.f, 0, 0});

    retrieveProcessedGeom(testGeom, vertexData);

    std::vector<float> expectedVertexData =
        {
            5.f, 0, -5.f,
            5.f, 0, 5.f,
            5.f, 10.f, -5.f,
            -5.f, 0, -5.f};

    EXPECT_NEAR_STDVEC(vertexData, expectedVertexData);
}

TEST(Geometry, translateGeom1)
{
    auto testGeom = Geometry();
    std::vector<float> vertexData =
        {
            0, 0, 0,
            10.f, 0, 0,
            0, 10.f, 0,
            0, 0, 10.f};

    ei::Vector3f translation = {-10.f, -10.f, -10.f};
    setupTestGeom(testGeom, vertexData);

    translateGeom(testGeom, translation);

    retrieveProcessedGeom(testGeom, vertexData);

    std::vector<float> expectedVertexData =
        {
            -10.f, -10.f, -10.f,
            0.f, -10.f, -10.f,
            -10.f, 0.f, -10.f,
            -10.f, -10.f, 0.f};

    EXPECT_NEAR_STDVEC(vertexData, expectedVertexData);
}

TEST(Geometry, transformGeom1)
{
    auto testGeom = Geometry();
    std::vector<float> vertexData =
        {
            1, 1, 1};

    ei::Transform3f transform = ei::Translation3f({10.f, 10.f, 10.f}) *
                                ei::AngleAxisf(TO_RAD(90), ei::Vector3f{0.f, 1.f, 0.f}) *
                                ei::Scaling(10.f, 10.f, 10.f);

    setupTestGeom(testGeom, vertexData);

    transformGeom(testGeom, transform);

    retrieveProcessedGeom(testGeom, vertexData);

    std::vector<float> expectedVertexData =
        {
            -110.f, 110.f, -110.f};

    EXPECT_NEAR_STDVEC(vertexData, expectedVertexData, 0.001);
}

TEST(Geometry, LoadGeometry)
{
    const aiScene *sceneCache = nullptr;

    std::filesystem::path assetDir = std::filesystem::absolute(kAssetDirectory);
    std::filesystem::path assetFile = assetDir / "cube_simple.obj";

    std::vector<const aiMesh *> meshes = loadAiMeshes(assetFile, &sceneCache);

    ASSERT_EQ(meshes.size(), 1);

    struct MeshData
    {
        std::vector<float> m_vertexData;
        std::vector<uint> m_edgeIdxs;
        std::vector<uint> m_faceIdxs;
    };
    std::map<std::string, MeshData> nameToMeshData;

    for (auto meshPtr : meshes)
    {
        MeshData &meshData = nameToMeshData[meshPtr->mName.data] = MeshData();

        meshData.m_vertexData.reserve(meshPtr->mNumVertices * 3);

        // std::cout << meshPtr->mNumVertices << std::endl;
        for (uint v = 0; v < meshPtr->mNumVertices; v++)
        {
            for (uint d = 0; d < 3; d++)
            {
                meshData.m_vertexData.push_back(meshPtr->mVertices[v][d]);
            }
        }

        // Assume faces have only 3 vertices
        meshData.m_faceIdxs.reserve(meshPtr->mNumFaces * 3);

        for (uint f = 0; f < meshPtr->mNumFaces; ++f)
        {
            const aiFace &face = meshPtr->mFaces[f];

            for (uint fi = 0; fi < face.mNumIndices; fi++)
            {
                meshData.m_faceIdxs.push_back(face.mIndices[fi]);
            }
        }
    }
    ASSERT_EQ(nameToMeshData.size(), 1);

    ASSERT_NE(nameToMeshData.find("Cube"), nameToMeshData.end());

    auto &meshData = nameToMeshData["Cube"];

    // printStdVecInStride(meshData.m_faceIdxs);

    ASSERT_EQ(meshData.m_vertexData.size(), 24);
    ASSERT_EQ(meshData.m_faceIdxs.size(), 36);

    std::vector<std::pair<std::string, Geometry *>> nameToGeometry =
        initGeometryFromAiMeshes<Geometry>(meshes);

    // std::vector<std::pair<std::string, Geometry *>> nameToGeometry =
    //     initGeometryFromAiMeshes<Geometry,
    //                              CuGlBufferSetter<float>,
    //                              CuGlBufferSetter<uint>,
    //                              CuGlBufferSetter<uint>>(meshes);

    Geometry *gPtr;
    for (auto &[name, geomPtr] : nameToGeometry)
        if (name == "Cube")
            gPtr = geomPtr;

    ASSERT_NE(gPtr, nullptr);

    Geometry &g = *gPtr;

    std::vector<uint> edgeIndices = deviceToContainer(
        g.d_edgeIdxBufferData,
        g.d_nEdgeIdxBufferElems);

    ASSERT_EQ(edgeIndices.size(), 18 * 2);

    for (auto [name, geometry] : nameToGeometry)
    {
        std::vector<float> vertexData;
        retrieveProcessedGeom(*geometry, vertexData);

        ASSERT_NE(nameToMeshData.find(name), nameToMeshData.end());

        ASSERT_EQ(vertexData, nameToMeshData[name].m_vertexData);
    }

    aiReleaseImport(sceneCache);
}

TEST(PBDGeometry, initializePBDParameters)
{
    const aiScene *sceneCache = nullptr;

    std::filesystem::path assetDir = std::filesystem::absolute(kAssetDirectory);
    std::filesystem::path assetFile = assetDir / "cube_simple.obj";

    std::vector<const aiMesh *> meshes = loadAiMeshes(assetFile, &sceneCache);

    DefaultCudaBufferSetter<uint> edgeSetter;
    std::vector<std::pair<std::string, PBDGeometry *>> nameToGeometry =
        initGeometryFromAiMeshes<PBDGeometry>(meshes, {}, edgeSetter, {});

    // std::cout << edgeSetter.m_data << std::endl;

    PBDGeometry *cubeGeom;
    for (auto &[name, geomPtr] : nameToGeometry)
        if (name == "Cube")
            cubeGeom = geomPtr;

    ASSERT_NE(cubeGeom, nullptr);

    uint *h_edgeIdxBufferData = new uint[cubeGeom->d_nEdgeIdxBufferElems];
    cutilSafeCall(hipMemcpy(h_edgeIdxBufferData,
                             cubeGeom->d_edgeIdxBufferData,
                             cubeGeom->d_nEdgeIdxBufferElems * sizeof(uint),
                             hipMemcpyDeviceToHost));

    // for (uint i = 0; i < cubeGeom->d_nEdgeIdxBufferElems; i++)
    // {
    //     std::cout << h_edgeIdxBufferData[i] << std::endl;
    // }

    uint fixedVertexIdx = 0;
    uint nFixedVertexIdx = 1;
    auto &g = *cubeGeom;
    auto &p = g.pbdData;
    initializePBDParameters(g, &fixedVertexIdx, nFixedVertexIdx);

    ASSERT_EQ(p.d_nFixedVertexIdxBufferElems, 1);

    std::vector<uint> fixedVertexIdxBufferData = deviceToContainer(
        p.d_fixedVertexIdxBufferData,
        p.d_nFixedVertexIdxBufferElems);

    ASSERT_EQ(fixedVertexIdxBufferData.size(), 1);
    ASSERT_EQ(fixedVertexIdxBufferData[0], 0);

    std::vector<float> distanceConstraintLengths = deviceToContainer(
        p.d_distanceConstraintLengthBufferData,
        p.d_nDistanceConstraintLengthBufferElems);
    std::vector<float> expectedDistanceConstraintLengths;

    ASSERT_EQ(distanceConstraintLengths.size(), 18);

    std::vector<float> vertexPositions = deviceToContainer(
        g.d_vertexPositionBufferData,
        g.d_nVertexPositionBufferElems);

    std::vector<uint> edgeIndices = deviceToContainer(
        g.d_edgeIdxBufferData,
        g.d_nEdgeIdxBufferElems);

    auto &getVector3fs = [](std::vector<float> v, size_t i)
    {
        return ei::Vector3f{{v[i], v[i + 1], v[i + 2]}};
    };
    auto &get2Vector3fs = [](std::vector<float> v, size_t i)
    {
        return std::pair<ei::Vector3f, ei::Vector3f>{
            {v[i], v[i + 1], v[i + 2]},
            {v[i + 3], v[i + 4], v[i + 5]}};
    };

    ASSERT_EQ(edgeIndices.size(), 18 * 2);
    ASSERT_EQ(vertexPositions.size() % 6, 0);

    for (size_t i = 0; i < edgeIndices.size(); i += 2)
    {
        uint idx1 = edgeIndices[i] * 3;
        uint idx2 = edgeIndices[i + 1] * 3;

        auto v1 = getVector3fs(vertexPositions, idx1);
        auto v2 = getVector3fs(vertexPositions, idx2);

        auto diff = v2 - v1;

        expectedDistanceConstraintLengths.push_back(length(diff));
    }

    ASSERT_EQ(distanceConstraintLengths, expectedDistanceConstraintLengths);

    std::vector<uint> distanceConstraintsIndices = deviceToContainer(
        p.d_distanceConstraintsIdxBufferData,
        p.d_nDistanceConstraintsIdxBufferElems);

    ASSERT_EQ(distanceConstraintsIndices, edgeIndices);

    std::vector<std::vector<uint>> distanceConstraintSets;
    std::vector<uint *> distanceConstraintSetsDevicePtrs = deviceToContainer(
        p.d_distanceConstraintSets,
        p.d_nDistanceConstraintSets);

    std::vector<uint> nDistanceConstraintsPerSet = deviceToContainer(
        p.d_nDistanceConstraintsPerSet,
        p.d_nDistanceConstraintSets);

    for (size_t i = 0; i < nDistanceConstraintsPerSet.size(); i++)
    {
        auto nDistanceConstraints = nDistanceConstraintsPerSet[i];
        distanceConstraintSets.push_back(
            deviceToContainer(
                distanceConstraintSetsDevicePtrs[i],
                nDistanceConstraints));
    }

    // Assert there is maximum one reference to a vertex index in each set
    for (const auto &dcSet : distanceConstraintSets)
    {
        std::map<uint, uint> idxToFrequency;

        for (auto &&idx : dcSet)
        {
            const auto [currEdgeItr, success] = idxToFrequency.insert({idx, 0});

            idxToFrequency[idx] += 1;
            ASSERT_EQ(currEdgeItr->second, 1);
        }
    }

    // std::cout << g.d_nVertexPositionBufferElems << std::endl;
    std::vector<float> vertexVelocitiesBuffer = deviceToContainer(
        p.d_vertexVelocitiesBufferData,
        g.d_nVertexPositionBufferElems);

    ASSERT_EQ(vertexVelocitiesBuffer.size(), 24);
    for (auto &&e : vertexVelocitiesBuffer)
    {
        ASSERT_EQ(e, 0.f);
    }

    size_t nVertices = g.d_nVertexPositionBufferElems / size_t(3);
    std::vector<float> vertexMassesBuffer = deviceToContainer(
        p.d_vertexMassesBufferData,
        nVertices);

    ASSERT_EQ(vertexMassesBuffer.size(), 8);
    for (auto &&e : vertexMassesBuffer)
    {
        ASSERT_EQ(e, 1.f);
    }
}

/*
TEST(Geometry, CuGlBufferSetter)
{
    CuGlBufferSetter<float, GL_ARRAY_BUFFER> vertexBufferSetter;

    std::vector<float> vertexData =
        {
            0, 0, 0,
            10.f, 0, 0,
            0, 10.f, 0,
            0, 0, 10.f};

    std::vector<float> expectedVertexData = {
        0, 0, 0,
        10.f, 0, 0,
        0, 10.f, 0,
        0, 0, 10.f};

    float *d_vertexData = nullptr;
    vertexBufferSetter.allocate(&d_vertexData, vertexData.size() * sizeof(float));
    vertexBufferSetter.copy(d_vertexData, &vertexData[0], vertexData.size());

    cutilSafeCall(
        hipMemcpy(devPtr, data, nElems * sizeof(T), hipMemcpyDeviceToHost));

    for (auto [name, geometry] : nameToGeometry)
    {
        std::vector<float> vertexData;
        retrieveProcessedGeom(*geometry, vertexData);

        ASSERT_NE(nameToMeshData.find(name), nameToMeshData.end());

        ASSERT_EQ(vertexData, nameToMeshData[name].m_vertexData);
    }
}
*/

TEST(testTest, test)
{
    EXPECT_EQ(true, true);
}
