#include "hip/hip_runtime.h"
#include "gtest/gtest.h"

#include <filesystem>
#include <map>
#include <sstream>
#include <string>
#include <vector>

#include "Camera.h"
#include "Geometry.cuh"
#include "LoadGeometry.cuh"

// Include GLM
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/type_ptr.hpp>

template <typename T, typename P = float>
bool _EXPECT_NEAR(T a, T b, P p)
{
    EXPECT_NEAR(a, b, p);
    T diff = std::abs(a - b);
    return p > diff && diff >= 0;
}

template <typename T, typename P = float>
void EXPECT_NEAR_STDVEC(T a, T b, P p = 0.00001)
{
    ASSERT_EQ(a.size(), b.size());
    bool val = true;
    for (int i = 0; i < a.size(); i++)
    {
        val &= _EXPECT_NEAR(a.data()[i], b.data()[i], p);
    }
    if (!val)
    {
        EXPECT_EQ(a, b);
    }
}

template <typename T, typename P = float>
void EXPECT_NEAR_MATRIX(T a, T b, P p = 0.00001)
{
    ASSERT_EQ(a.rows(), b.rows());
    ASSERT_EQ(a.cols(), b.cols());
    bool val = true;
    for (int i = 0; i < a.rows() * a.cols(); i++)
    {
        val &= _EXPECT_NEAR(a.data()[i], b.data()[i], p);
    }
    if (!val)
    {
        EXPECT_EQ(a, b);
    }
}

void print_glm_mat4(glm::mat4 &mat)
{
    for (int r = 0; r < 4; r++)
    {
        for (int c = 0; c < 4; c++)
        {
            fprintf(stdout, "%7.1f", mat[r][c]);
        }
        fprintf(stdout, "\n");
    }
}

TEST(EigenTests, ProjectionAndViewTest)
{
    float radians_fov = 45.0f;
    float windowWidth = 1920.f;
    float windowHeight = 1080.f;
    float fov = 45.f;
    float far = 100.0f;
    float near = 0.1f;

    glm::mat4 glm_projection = glm::perspective(
        glm::radians(radians_fov), windowWidth / windowHeight, near, far);
    glm::mat4 glm_view = glm::lookAt(
        glm::vec3(0, 0, 30), // Camera is at (4,3,-3), in World Space
        glm::vec3(0, 0, 0),  // and looks at the origin
        glm::vec3(0, 1, 0)   // Head is up (set to 0,-1,0 to look upside-down)
    );

    glm::mat4 glm_PV = glm_projection * glm_view;

    ei::Matrix4f ei_projection;
    ei::Matrix4f ei_view;

    ei_utils::setProjMat(ei_projection, windowWidth, windowHeight, TO_RAD(fov), far, near);
    ei_utils::setLookAt(ei_view, {0, 0, 30}, {0, 0, 0}, {0, 1, 0});

    // ei::Matrix4f ei_PV = ei_projection.transpose() * ei_view.transpose();
    ei::Matrix4f ei_PV = ei_projection * ei_view;

    ASSERT_EQ(TO_RAD(fov), glm::radians(radians_fov));

    for (int i = 0; i < 16; i++)
    {
        EXPECT_NEAR(glm::value_ptr(glm_projection)[i], ei_projection.data()[i], 0.1);
        EXPECT_NEAR(glm::value_ptr(glm_view)[i], ei_view.data()[i], 0.1);
        EXPECT_NEAR(glm::value_ptr(glm_PV)[i], ei_PV.data()[i], 0.1);
    }

    //    std::cout << "PROJECTION MATRICES" << std::endl;
    //    std::cout << ei_projection << std::endl;
    //    print_glm_mat4(glm_projection);
    //
    //    std::cout << "VIEW MATRICES" << std::endl;
    //    std::cout << ei_view << std::endl;
    //    print_glm_mat4(glm_view);
    //
    //    std::cout << "PROJECTION VIEW MATRICES" << std::endl;
    //    std::cout << ei_PV.transpose() << std::endl;
    //    print_glm_mat4(glm_PV);
    //    //std::cout << glm::to_string(glm_projection) << std::endl;
    //
    //
    //    for (int i=0; i<16; i++)
    //    {
    //        auto* p = &glm_projection[0][0];
    //        assert(p[i] == ei_projection.data()[i]);
    //        auto* v = &glm_view[0][0];
    //        assert(v[i] == ei_view.data()[i]);
    //        auto* pv = &glm_PV[0][0];
    //        assert(pv[i] == ei_PV.data()[i]);
    //    }
    //
    //    for (int r=0; r<4; r++)
    //        for (int c=0; c<4; c++)
    //        {
    //            assert(glm_projection[r][c] == ei_projection(r, c));
    //            assert(glm_view[r][c] == ei_view(r, c));
    //            assert(glm_PV[r][c] == ei_PV(r, c));
    //        }
}

using MapVector3f = Eigen::Map<ei::Vector3f>;

void setupTestGeom(Geometry &testGeom,
                   std::vector<float> &vertexData)
{
    testGeom.d_nVertexPositionBufferElems = vertexData.size();

    size_t bufferBytesSize = testGeom.d_nVertexPositionBufferElems *
                             sizeof(float);

    hipMalloc((void **)&testGeom.d_vertexPositionBufferData,
               bufferBytesSize);

    hipMemcpy(testGeom.d_vertexPositionBufferData,
               &vertexData.data()[0],
               bufferBytesSize,
               hipMemcpyHostToDevice);
}

void retrieveProcessedGeom(Geometry &testGeom,
                           std::vector<float> &vertexData)
{
    vertexData.resize(testGeom.d_nVertexPositionBufferElems);
    size_t bufferBytesSize = testGeom.d_nVertexPositionBufferElems *
                             sizeof(float);
    hipMemcpy(static_cast<void *>(&vertexData.data()[0]),
               static_cast<const void *>(testGeom.d_vertexPositionBufferData),
               bufferBytesSize,
               hipMemcpyDeviceToHost);

    hipFree(testGeom.d_vertexPositionBufferData);
}

TEST(Geometry, scaleGeom1)
{
    auto testGeom = Geometry();
    std::vector<float> vertexData =
        {
            0, 0, 0,
            1, 0, 0,
            0, 1, 0,
            0, 0, 1};

    ei::Vector3f scale = {2.5f, 5.0f, 10.0f};
    setupTestGeom(testGeom, vertexData);

    scaleGeom(testGeom, scale);

    retrieveProcessedGeom(testGeom, vertexData);

    std::vector<float> expectedVertexData =
        {
            0, 0, 0,
            2.5f, 0, 0,
            0, 5.0f, 0,
            0, 0, 10.0f};

    ASSERT_EQ(vertexData, expectedVertexData);
}

TEST(Geometry, scaleGeom2)
{
    auto testGeom = Geometry();
    std::vector<float> vertexData =
        {
            0, 0, 0,
            1, 0, 0,
            0, 1, 0,
            0, 0, 1};

    ei::Vector3f scale = {5.f, 5.f, 5.f};
    setupTestGeom(testGeom, vertexData);

    scaleGeom(testGeom, scale);

    retrieveProcessedGeom(testGeom, vertexData);

    std::vector<float> expectedVertexData =
        {
            0, 0, 0,
            5.f, 0, 0,
            0, 5.f, 0,
            0, 0, 5.f};

    ASSERT_EQ(vertexData, expectedVertexData);
}

TEST(Geometry, scaleGeom3)
{
    auto testGeom = Geometry();
    std::vector<float> vertexData =
        {
            0, 0, 0,
            10.f, 0, 0,
            0, 10.f, 0,
            0, 0, 10.f};

    ei::Vector3f scale = {2.f, 2.f, 2.f};
    ei::Vector3f pivot = {5.f, 5.f, 5.f};
    setupTestGeom(testGeom, vertexData);

    scaleGeom(testGeom, scale, pivot);

    retrieveProcessedGeom(testGeom, vertexData);

    std::vector<float> expectedVertexData =
        {
            -5, -5, -5,
            15, -5, -5,
            -5, 15, -5,
            -5, -5, 15};

    ASSERT_EQ(vertexData, expectedVertexData);
}

TEST(Geometry, rotateGeom1)
{
    auto testGeom = Geometry();
    std::vector<float> vertexData =
        {
            0, 0, 0,
            10.f, 0, 0,
            0, 10.f, 0,
            0, 0, 10.f};

    ei::Vector3f axis = {0, 1.f, 0};

    setupTestGeom(testGeom, vertexData);

    rotateGeom(testGeom, axis, TO_RAD(90), {0, 0, 0});

    retrieveProcessedGeom(testGeom, vertexData);

    std::vector<float> expectedVertexData =
        {
            0, 0, 0,
            0, 0, 10.f,
            0, 10.f, 0,
            -10.f, 0, 0};

    EXPECT_NEAR_STDVEC(vertexData, expectedVertexData);
}

TEST(Geometry, rotateGeom2)
{
    auto testGeom = Geometry();
    std::vector<float> vertexData =
        {
            0, 0, 0,
            10.f, 0, 0,
            0, 10.f, 0,
            0, 0, 10.f};

    ei::Vector3f axis = {0, 1.f, 0};

    setupTestGeom(testGeom, vertexData);

    rotateGeom(testGeom, axis, TO_RAD(90), {5.f, 0, 0});

    retrieveProcessedGeom(testGeom, vertexData);

    std::vector<float> expectedVertexData =
        {
            5.f, 0, -5.f,
            5.f, 0, 5.f,
            5.f, 10.f, -5.f,
            -5.f, 0, -5.f};

    EXPECT_NEAR_STDVEC(vertexData, expectedVertexData);
}

TEST(Geometry, translateGeom1)
{
    auto testGeom = Geometry();
    std::vector<float> vertexData =
        {
            0, 0, 0,
            10.f, 0, 0,
            0, 10.f, 0,
            0, 0, 10.f};

    ei::Vector3f translation = {-10.f, -10.f, -10.f};
    setupTestGeom(testGeom, vertexData);

    translateGeom(testGeom, translation);

    retrieveProcessedGeom(testGeom, vertexData);

    std::vector<float> expectedVertexData =
        {
            -10.f, -10.f, -10.f,
            0.f, -10.f, -10.f,
            -10.f, 0.f, -10.f,
            -10.f, -10.f, 0.f};

    EXPECT_NEAR_STDVEC(vertexData, expectedVertexData);
}

TEST(Geometry, transformGeom1)
{
    auto testGeom = Geometry();
    std::vector<float> vertexData =
        {
            1, 1, 1};

    ei::Transform3f transform = ei::Translation3f({10.f, 10.f, 10.f}) *
                                ei::AngleAxisf(TO_RAD(90), ei::Vector3f{0.f, 1.f, 0.f}) *
                                ei::Scaling(10.f, 10.f, 10.f);

    setupTestGeom(testGeom, vertexData);

    transformGeom(testGeom, transform);

    retrieveProcessedGeom(testGeom, vertexData);

    std::vector<float> expectedVertexData =
        {
            -110.f, 110.f, -110.f};

    EXPECT_NEAR_STDVEC(vertexData, expectedVertexData, 0.001);
}

TEST(Geometry, LoadGeometry)
{
    const aiScene *sceneCache = nullptr;

    std::filesystem::path assetFile(__FILE__);

    assetFile = std::filesystem::absolute(
        assetFile.parent_path() / ".." / "assets" / "cube_simple.obj");
    std::cout << assetFile << std::endl;

    std::vector<const aiMesh *> meshes = loadAiMeshes(assetFile, &sceneCache);

    ASSERT_EQ(meshes.size(), 1);

    struct MeshData
    {
        std::vector<float> m_vertexData;
        std::vector<uint> m_faceIdxs;
    };
    std::map<std::string, MeshData> nameToMeshData;

    for (auto meshPtr : meshes)
    {
        MeshData &meshData = nameToMeshData[meshPtr->mName.data] = MeshData();

        meshData.m_vertexData.reserve(meshPtr->mNumVertices * 3);

        std::cout << meshPtr->mNumVertices << std::endl;
        for (uint v = 0; v < meshPtr->mNumVertices; v++)
        {
            for (uint d = 0; d < 3; d++)
            {
                meshData.m_vertexData.push_back(meshPtr->mVertices[v][d]);
            }
        }

        // Assume faces have only 3 vertices
        meshData.m_faceIdxs.reserve(meshPtr->mNumFaces * 3);

        for (uint f = 0; f < meshPtr->mNumFaces; ++f)
        {
            const aiFace &face = meshPtr->mFaces[f];

            for (uint fi = 0; fi < face.mNumIndices; fi++)
            {
                meshData.m_faceIdxs.push_back(face.mIndices[fi]);
            }
        }
    }
    ASSERT_EQ(nameToMeshData.size(), 1);

    ASSERT_NE(nameToMeshData.find("Cube"), nameToMeshData.end());

    auto &meshData = nameToMeshData["Cube"];
    printStdVecInStride(meshData.m_faceIdxs);
    ASSERT_EQ(meshData.m_vertexData.size(), 24);
    ASSERT_EQ(meshData.m_faceIdxs.size(), 36);

    std::vector<std::pair<std::string, Geometry *>> nameToGeometry =
        initGeometryFromAiMeshes<Geometry>(meshes);

    // std::vector<std::pair<std::string, Geometry *>> nameToGeometry =
    //     initGeometryFromAiMeshes<Geometry,
    //                              CuGlBufferSetter<float>,
    //                              CuGlBufferSetter<uint>,
    //                              CuGlBufferSetter<uint>>(meshes);

    for (auto [name, geometry] : nameToGeometry)
    {
        std::vector<float> vertexData;
        retrieveProcessedGeom(*geometry, vertexData);

        ASSERT_NE(nameToMeshData.find(name), nameToMeshData.end());

        ASSERT_EQ(vertexData, nameToMeshData[name].m_vertexData);
    }

    aiReleaseImport(sceneCache);
}
/*
TEST(Geometry, CuGlBufferSetter)
{
    CuGlBufferSetter<float, GL_ARRAY_BUFFER> vertexBufferSetter;

    std::vector<float> vertexData =
        {
            0, 0, 0,
            10.f, 0, 0,
            0, 10.f, 0,
            0, 0, 10.f};

    std::vector<float> expectedVertexData = {
        0, 0, 0,
        10.f, 0, 0,
        0, 10.f, 0,
        0, 0, 10.f};

    float *d_vertexData = nullptr;
    vertexBufferSetter.allocate(&d_vertexData, vertexData.size() * sizeof(float));
    vertexBufferSetter.copy(d_vertexData, &vertexData[0], vertexData.size());

    cutilSafeCall(
        hipMemcpy(devPtr, data, nElems * sizeof(T), hipMemcpyDeviceToHost));

    for (auto [name, geometry] : nameToGeometry)
    {
        std::vector<float> vertexData;
        retrieveProcessedGeom(*geometry, vertexData);

        ASSERT_NE(nameToMeshData.find(name), nameToMeshData.end());

        ASSERT_EQ(vertexData, nameToMeshData[name].m_vertexData);
    }
}
*/

TEST(testTest, test)
{
    EXPECT_EQ(true, true);
}
