#include "hip/hip_runtime.h"
#include "CuGlGeometry.cuh"

void __global__ setBufferVals(float setNum, float *d_bufferPtr,
                              int bufferSize)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < bufferSize)
        d_bufferPtr[idx] = idx;
}

void __global__ addToBufferVertex(ei::Vector3f setNum, float *d_bufferPtr,
                                  int bufferSize)
{
    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 3;
    int idx_x = idx;
    int idx_y = idx + 1;
    int idx_z = idx + 2;

    if (idx_x < bufferSize)
        d_bufferPtr[idx_x] += setNum.x();
    if (idx_y < bufferSize)
        d_bufferPtr[idx_y] += setNum.y();
    if (idx_z < bufferSize)
        d_bufferPtr[idx_z] += setNum.z();
}

void drawGeom(CuGlGeometry const &geom, Eigen::Matrix4f &cameraMat)
{

    // std::cout<<"DEBUG 1"<<std::endl;
    glUseProgram(*geom.monoColourShader);
    // std::cout<<"DEBUG 1.25"<<std::endl;
    GLuint mvpID = glGetUniformLocation(*geom.monoColourShader, "MVP");
    // std::cout<<"DEBUG 1.5"<<std::endl;
    glUniformMatrix4fv(mvpID, 1, GL_FALSE, cameraMat.data());

    // std::cout<<"DEBUG 2"<<std::endl;
    GLuint baseColID =
        glGetUniformLocation(*geom.monoColourShader, "base_colour");
    glUniform3fv(baseColID, 1, geom.baseColour.data());

    // std::cout<<"DEBUG 3"<<std::endl;
    glEnableVertexAttribArray(0);
    glBindBuffer(GL_ARRAY_BUFFER, geom.buffer.gl_VBO);
    glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0, NULL);
    glDrawArrays(GL_LINES, 0, geom.buffer.d_bufferSize / 3);
    glDisableVertexAttribArray(0);
}

void translateGeom(CuGlGeometry &geom, const ei::Vector3f &setNum)
{

    auto &d_pBuffer = geom.buffer.d_pBuffer;
    auto &cugl_pVBO = geom.buffer.cugl_pVBO;
    size_t bufferSize = geom.buffer.d_bufferSize;
    size_t bufferSizeBytes;

    // Map buffer object
    cutilSafeCall(hipGraphicsMapResources(1, &cugl_pVBO));

    // Get pointer to use, not sure if possible to use outside of mapped scope
    cutilSafeCall(hipGraphicsResourceGetMappedPointer(
        (void **)&d_pBuffer, &bufferSizeBytes, cugl_pVBO));

    addToBufferVertex<<<1, static_cast<int>((float)bufferSize / 3.f)>>>(
        setNum, d_pBuffer, bufferSize);

    // Unmap buffer object
    cutilSafeCall(hipGraphicsUnmapResources(1, &cugl_pVBO));
}