#include "hip/hip_runtime.h"

#include "Utils/ImGUI.h"
#include "Utils/Mesh.h"

#include "Geometry/CuGlBuffer.cuh"
#include "Geometry/CuGlGeometry.cuh"
#include "Geometry/Geometry.cuh"
#include "Geometry/LoadGeometry.cuh"
#include "Ui/GLFWState.h"

#include "Sim/PBD/PBDSolver.cuh"

#include "Viewer/Camera.h"
#include "Viewer/GeometryViewer.h"
#include "Viewer/MonoColourGLShader.h"
#include "Viewer/PlaneGLData.h"

// struct CUDA_GL_state{
//     int deviceCount;
//     hipDeviceProp_t deviceProp;
//     std::vector<int> dev;
// };

// CUDA_GL_state initCUDAGLContext(){
//     CUDA_GL_state state;
//     hipGetDeviceCount(&state.deviceCount);
//     for (int i=0; i<state.deviceCount; i++) {
//         hipGetDeviceProperties(&state.deviceProp, state.dev[i]);
//     }
//     // Set Gl device to 1st device (in case of multiple GPUs ?)
//     cudaGLSetGLDevice(state.dev[0]);
//     return state;
// }

#define STR(s) #s
#define XSTR(s) STR(s)

static constexpr char *kAssetDirectory = XSTR(ASSETS_DIRECTORY);

template <typename T>
std::vector<T> deviceToContainer(T *d_ptr, size_t nElems)
{
    std::vector<T> result(nElems);

    cutilSafeCall(hipMemcpy(result.data(),
                             d_ptr,
                             nElems * sizeof(T),
                             hipMemcpyDeviceToHost));
    return result;
}

void ImGuiHelloWorld(bool showDemoWindow, ImVec4 &clearColor)
{
    static float f = 0.0f;
    static int counter = 0;

    ImGui::Begin("Hello, world!"); // Create a window called "Hello, world!" and
                                   // append into it.

    ImGui::Text("This is some useful text."); // Display some text (you can use a
                                              // format strings too)
    ImGui::Checkbox(
        "Demo Window",
        &showDemoWindow); // Edit bools storing our window open/close state
    ImGui::Checkbox("Another Window", &showDemoWindow);

    ImGui::SliderFloat("float", &f, 0.0f,
                       1.0f); // Edit 1 float using a slider from 0.0f to 1.0f
    ImGui::ColorEdit3("clear color",
                      (float *)&clearColor); // Edit 3 floats representing a color

    if (ImGui::Button("Button")) // Buttons return true when clicked (most widgets
                                 // return true when edited/activated)
        counter++;
    ImGui::SameLine();
    ImGui::Text("counter = %d", counter);

    ImGui::Text("Application average %.3f ms/frame (%.1f FPS)",
                1000.0f / ImGui::GetIO().Framerate, ImGui::GetIO().Framerate);
    ImGui::End();
}

int main(int argv, char **args)
{

    fprintf(stdout, "Start of main.\n");

    //=====================================GLEW/GLFW SETUP=======================================

    int glMajorVersion = 4;
    int glMinorVersion = 6;
    // TODO: Change to 450
    const char *glslVersion = "#version 330";

    fprintf(stdout, "GLEW/GLFW SETUP\n");

    // Initialize GLFW
    auto glfwState = setupGLFW(glMajorVersion, glMinorVersion);

    // Create a GLFW window
    auto mainWindow = createWindow("Main Window");
    auto windowAddedResult = addWindow(glfwState, mainWindow);
    glfwMakeContextCurrent(mainWindow->m_glfwWindow);

    fprintf(stdout, "Adding window result: %i\n", windowAddedResult);
    fprintf(stdout, "m_windows size: %i\n", (int)glfwState.m_windows.size());

    if (glewInit() != GLEW_OK)
    {
        fprintf(stderr, "Failed to initialize GLEW.\n");
        return EXIT_FAILURE;
    }

    //=====================================IMGUI SETUP==========================================

    // Setup Dear ImGui context
    IMGUI_CHECKVERSION();
    ImGui::CreateContext();
    ImGuiIO &io = ImGui::GetIO();
    (void)io;
    // io.ConfigFlags |= ImGuiConfigFlags_NavEnableKeyboard;     // Enable
    // Keyboard Controls io.ConfigFlags |= ImGuiConfigFlags_NavEnableGamepad; //
    // Enable Gamepad Controls

    // Setup Dear ImGui style
    ImGui::StyleColorsDark();
    // ImGui::StyleColorsLight();

    // Setup Platform/Renderer backends
    ImGui_ImplGlfw_InitForOpenGL(mainWindow->m_glfwWindow, true);
    ImGui_ImplOpenGL3_Init(glslVersion);

    bool showDemoWindow = true;

    //=====================================CUDA SETUP==============================================
    // Initialize CUDA context (on top of the GL context)

    fprintf(stdout, "CUDA SETUP\n");

    // hipSetDevice(0);
    // cudaGLSetGLDevice(0);

    //=====================================OPENGL SETUP============================================

    fprintf(stdout, "OPENGL SETUP\n");

    ImVec4 clearColor = {0.5f, 0.5f, 0.5f, 1.f};

    glClearColor(clearColor.x * clearColor.w, clearColor.y * clearColor.w,
                 clearColor.z * clearColor.w, clearColor.w);
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glEnable(GL_DEPTH_TEST);
    // glDepthFunc(GL_LESS);

    // glfwSwapBuffers(mainWindow->m_glfwWindow);

#ifndef NDEBUG
    // During init, enable debug output
    glEnable(GL_DEBUG_OUTPUT);
    glDebugMessageCallback(GLDebugMessageCallback, 0);
#endif

    //=====================================CAMERA SETUP============================================

    fprintf(stdout, "CAMERA SETUP\n");

    auto camera = Camera(mainWindow->m_windowWidth, mainWindow->m_windowHeight);

    // yawCamera(camera, TO_RAD(-45.f));
    // pitchCamera(camera, TO_RAD(-45.f));

    updateCamera(camera);

    using KEY_ID = int;
    // std::map<KEY_ID, Camera::Actions> cameraKeyToAction = {
    //     {GLFW_KEY_UP, Camera::ORBIT_UP},
    //     {GLFW_KEY_LEFT, Camera::ORBIT_LEFT},
    //     {GLFW_KEY_RIGHT, Camera::ORBIT_RIGHT},
    //     {GLFW_KEY_DOWN, Camera::ORBIT_DOWN}
    // };
    std::map<KEY_ID, Camera::Actions> cameraKeyToAction = {
        {GLFW_KEY_W, Camera::ORBIT_UP},
        {GLFW_KEY_A, Camera::ORBIT_LEFT},
        {GLFW_KEY_D, Camera::ORBIT_RIGHT},
        {GLFW_KEY_S, Camera::ORBIT_DOWN},

        {GLFW_KEY_UP, Camera::PAN_UP},
        {GLFW_KEY_DOWN, Camera::PAN_DOWN},
        {GLFW_KEY_LEFT, Camera::PAN_LEFT},
        {GLFW_KEY_RIGHT, Camera::PAN_RIGHT}};

    //=====================================SHADER SETUP============================================

    fprintf(stdout, "SHADER SETUP\n");

    GLuint monoColourShader = compileMonoColourShaderProgram();
    if (monoColourShader == 0)
    {
        fprintf(stderr, "Shader setup failed.\n");
        return EXIT_FAILURE;
    }

    checkGLError();

    //=====================================SCENE DATA LOAD=========================================
    const aiScene *sceneCache = nullptr;

    std::filesystem::path assetDir = std::filesystem::absolute(kAssetDirectory);
    // std::filesystem::path assetFile = assetDir / "PantherBoss" / "PAN.obj";
    std::filesystem::path assetFile = assetDir / "cube_simple.obj";

    std::cout << assetFile << std::endl;

    std::vector<const aiMesh *> meshes = loadAiMeshes(assetFile, &sceneCache);

    //=====================================MESH DATA SETUP=========================================

    fprintf(stdout, "MESH SETUP\n");

    // Create center of world grid plain
    std::vector<float> gridPlaneVertexData;
    ei::Vector3f cubeGridOrigin = {0.f, 0.f, 0.f};

    // generateSquare(gridPlaneVertexData,
    //                     ei::Vector3f(0.f, 0.f, 0.f),
    //                     1.f,
    //                     {Dim::X, Dim::Z});
    generateSquarePlane(gridPlaneVertexData, ei::Vector3f(0.f, 0.f, 0.f), 1.f,
                        {Dim::X, Dim::Z}, ei::Vector2ui(10, 10));
    // generateCubeGrid(gridPlaneVertexData,
    //                  cubeGridOrigin,
    //                  1.f,
    //                  ei::Vector3ui(10, 10, 10));
    float cubeGridTranslate[4] = {
        cubeGridOrigin[0],
        cubeGridOrigin[1],
        cubeGridOrigin[2],
        0,
    };

    // PlaneGLData gridPlane(&gridPlaneVertexData, &monoColourShader);
    // initPlaneVAO(gridPlane);

    CuGlGeometry gridPlaneCu(&gridPlaneVertexData, &monoColourShader);

    CuGlBufferSetter<float> vertexBufferSetter;
    CuGlBufferSetter<uint, GL_ELEMENT_ARRAY_BUFFER> indexBufferSetter;

    std::vector<std::pair<std::string, PBDGeometry *>> nameToGeometry =
        initGeometryFromAiMeshes<PBDGeometry>(meshes, vertexBufferSetter, {},
                                              indexBufferSetter);

    PBDGeometry &cudaCube = *(nameToGeometry[0].second);

    std::vector<uint> fixedVertices{0};

    initializePBDParameters(cudaCube, fixedVertices.data(),
                            static_cast<uint>(fixedVertices.size()));

    // clang-format off
    const auto &mapAndSyncCuGlBuffers = [&]()
    {
        vertexBufferSetter.mapAndSync(&cudaCube.d_vertexPositionBufferData);
        indexBufferSetter.mapAndSync(&cudaCube.d_edgeIdxBufferData);
    };

    const auto &unMapCuGlBuffers = [&]()
    {
        vertexBufferSetter.unMap();
        indexBufferSetter.unMap();
    };
    // clang-format on

    GeometryViewer cubeViewer{};

    initGeometryViewer(cubeViewer,
                       vertexBufferSetter.m_nElements,
                       vertexBufferSetter.m_glBufferId,
                       indexBufferSetter.m_nElements,
                       indexBufferSetter.m_glBufferId,
                       &monoColourShader);

    WorldProperties props;

    //=====================================MAIN LOOP===============================================
    int frame = 0;

    while (!shouldQuit(glfwState))
    {
        frame++;
        for (auto window : glfwState.m_windows)
        {
            glfwMakeContextCurrent(window->m_glfwWindow);

            glClearColor(clearColor.x * clearColor.w, clearColor.y * clearColor.w,
                         clearColor.z * clearColor.w, clearColor.w);
            glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

            // Handle interaction
            // TODO: handle better than.. this?
            glfwPollEvents();

            for (auto [key, action] : cameraKeyToAction)
            {
                int state = glfwGetKey(window->m_glfwWindow, key);
                if (state == GLFW_PRESS)
                {
                    // std::cout<<"Key pressed: "<<key<<std::endl;
                    moveCamera(camera, action);
                }
            }

            if (window->m_yScroll > 0)
            {
                moveCamera(camera, Camera::ZOOM_IN);
                window->m_yScroll = 0;
            }
            else if (window->m_yScroll < 0)
            {
                moveCamera(camera, Camera::ZOOM_OUT);
                window->m_yScroll = 0;
            }

            /*
            //translateGeom();
            */

            // Start ImGui frame
            ImGui_ImplOpenGL3_NewFrame();
            ImGui_ImplGlfw_NewFrame();
            ImGui::NewFrame();

            bool showDemoWindow = true;
            {
                // ImGui::ShowDemoWindow(&showDemoWindow);
            }

            ImGuiHelloWorld(showDemoWindow, clearColor);

            //{
            //
            //    bool show_another_window = true;
            //    ImGui::Begin("Grid Translation", &show_another_window);
            //    ImGui::InputFloat3("input float3", cubeGridTranslate);
            //    ImGui::SameLine();
            //    if (ImGui::Button("Translate"))
            //        translateGeom(gridPlaneCu,
            //                      {cubeGridTranslate[0],
            //                       cubeGridTranslate[1],
            //                       cubeGridTranslate[2]});
            //    ImGui::End();
            //}

            {

                bool show_another_window = true;
                ImGui::Begin("Cube Translation", &show_another_window);
                ImGui::InputFloat3("input float3", cubeGridTranslate);
                ImGui::SameLine();
                if (ImGui::Button("Translate"))
                {
                    ei::Vector3f translation = {cubeGridTranslate[0],
                                                cubeGridTranslate[1],
                                                cubeGridTranslate[2]};
                    std::cout << translation << std::endl;

                    mapAndSyncCuGlBuffers();
                    translateGeom(cudaCube, translation);
                    unMapCuGlBuffers();
                }
                ImGui::End();
            }

            ImGui::Render();

            // Camera update
            updateCamera(camera);
            ei::Matrix4f cameraVP = camera.projMat * camera.viewMat;

            mapAndSyncCuGlBuffers();

            std::vector<float> positionsPreTranslate = deviceToContainer(
                cudaCube.d_vertexPositionBufferData,
                cudaCube.d_nVertexPositionBufferElems);

            std::cout << positionsPreTranslate << std::endl;
            applyExternalForces(cudaCube, props);
            runPBDSolver(cudaCube);

            unMapCuGlBuffers();
            std::vector<float> positionsPostTranslate = deviceToContainer(
                cudaCube.d_vertexPositionBufferData,
                cudaCube.d_nVertexPositionBufferElems);
            std::cout << positionsPostTranslate << std::endl;

            // Draw geometry
            // updatePlaneVBO(gridPlane);
            // drawPlane(gridPlane, cameraVP);
            checkGLError();

            drawGeometryViewer(cubeViewer, cameraVP);
            checkGLError();

            drawGeom(gridPlaneCu, cameraVP);
            checkGLError();

            // Overlay imgui stuff
            ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());

            glfwSwapBuffers(window->m_glfwWindow);
        }
    }

    teardown(glfwState);
    return EXIT_SUCCESS;
}